#include "hip/hip_runtime.h"
﻿/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* NVIDIA Corporation is strictly prohibited.
*
* Please refer to the applicable NVIDIA end user license agreement (EULA)
* associated with this source code for terms and conditions that govern
* your use of this NVIDIA software.
*
*/
#include <iostream>
#include <fstream>
#include <dos.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"
#include <cstdlib>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <iostream>
#include <fstream>

#define SPHERES 5
#define DIM 1200
#define DIMTH 768
#define NUMTEST 200

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 1024
#define LIGHTX 380
#define LIGHTY 768
#define LIGHTZ 20
#define NUMREFLECTION 3

using namespace std;
__device__ double pow(double x,double y){

return x*x;
}



__device__ int controlSide(int indice,int *matrix, int radiusAVG){
	int countAVG = 0;
	if(indice + radiusAVG <= DIM * DIM && indice - radiusAVG >= 0){
			//controllo i lati della posizione
			
			for(int i = 1; i < radiusAVG; i++){

				int dx = i;
				int sx = -i;

				countAVG+=matrix[indice + dx];
				
				countAVG+=matrix[indice + sx];
				


			}

		}


	return countAVG;
}

struct Sphere {
	float   r, b, g;
	float   radius;
	float   x, y, z;
	__device__ float hit(float ox, float oy, float *n) {
		float dx = ox - x;
		float dy = oy - y;
		if (dx*dx + dy*dy < radius*radius) {
			float dz = sqrtf(radius*radius - dx*dx - dy*dy);
			*n = dz / sqrtf(radius * radius);
			//*n = 1.0;
			return dz + z;
		
		}
		return -INF;
		
	}
};


struct Point{
	int x;
	int y;
	int z;
	double d;
	int j;

};

__device__ double dot3(double a[], double b[]){

	return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}



__device__ void reflection(Point maxp, double dirx, double diry,double dirz, Sphere * s,unsigned char *ptr, int *matrix, Point *pointmatrix){
	
	int indice;
	Point points[SPHERES];

	for(int k=0; k<NUMREFLECTION;k++){

			int indx = maxp.j;

			double nX = maxp.x -  s[indx].x;
			double nY = maxp.y -  s[indx].y;
			double nZ = maxp.z -  s[indx].z;

			double a[] = {nX,nY,nZ};
			double b[] = {dirx,diry,dirz};

			float fact = 2*(dot3(a,b));
			double c[] = {fact*a[0],fact*a[1],fact*a[2]};
			double ray[] = {c[0] - b[0],c[1]-b[1],c[2]-b[2]};

			dirx=ray[0];
			diry=ray[1];
			dirz=ray[2];

		for (int j = 0; j<SPHERES; j++) {
			
				
				Point p;
				p.x=-1;
				p.y=-1;
				p.z=-1;
				p.d=-1;
				p.j=-1;

				//printf("Intersezione con sfera %d punti x=%f , y=%f z=%f\n",j,dirx,diry,dirz);

				float A = pow(dirx,2.0)+pow(diry,2.0)+pow(dirz,2.0);
				float B = 2.0 * (((LIGHTX-s[j].x)*dirx + (LIGHTY-s[j].y)*diry + (LIGHTZ - s[j].z )*dirz));
				float C = (pow((LIGHTX-s[j].x),2) + pow((LIGHTY-s[j].y),2) + pow((LIGHTZ-s[j].z),2)) - pow(s[j].radius,2);
				float D = B*B-4*A*C;
				

				if(D > 0.0){

					double t0 = ((-B) + sqrtf(D))/(2*A);
					double t1 = ((-B) - sqrtf(D))/(2*A);

					double ix0 = LIGHTX + (t0*dirx);
					double iy0 = LIGHTY + (t0*diry);
					double iz0 = LIGHTZ + (t0*dirz);

					double ix1 = LIGHTX + (t1*dirx);
					double iy1 = LIGHTY + (t1*diry);
					double iz1 = LIGHTZ + (t1*dirz);					

					double d0 = sqrt(pow((ix0-LIGHTX),2)+pow((iy0-LIGHTY),2)+pow((iz0-LIGHTZ),2));
					double d1 = sqrt(pow((ix1-LIGHTX),2)+pow((iy1-LIGHTY),2)+pow((iz1-LIGHTZ),2));

					int ix;
					int iy;
					int iz;
					double d;

					if(t0>0 && t1>0){
						if(t0<t1){
							ix = ix0;
							iy = iy0;
							iz = iz0;
							d=t0;
						}else{
							ix = ix1;
							iy = iy1;
							iz = iz1;
							d=t1;
						}
					}else{
						if(t0<0 && t1>0){
							ix = ix1;
							iy = iy1;
							iz = iz1;
							d=t1;
						}else{
							if(t1<0 && t0>0){
								ix = ix0;
								iy = iy0;
								iz = iz0;
								d=t0;
							}
						}
					}

					if(ix<0 || iy<0 || iz <0 || ix>DIM || iy>DIM || iz > DIM)
						continue;
				
					//hostmatrix[iy][ix]=1;

					p.x=ix;
					p.y=iy;
					p.z=iz;
					p.d=d;
					p.j=j;
					points[j]=p;

					
					//count++;
					

				}
			
				if(D < 0.0){
					points[j]=p;
				}

			}

			maxp.d=4294967295;

			for (int j = 0; j<SPHERES; j++) {


				if(points[j].x == -1 || (points[j].x == 0 && points[j].y == 0 && points[j].z == 0 && points[j].d == 0) || points[j].d<0 )
					continue;			

				if(maxp.d>points[j].d){
					maxp=points[j];
				}

			}

			if(maxp.d==4294967295)
				continue;

			indice = (DIM * maxp.y)+maxp.x;
			atomicAdd(&matrix[indice], (matrix[indice] +=1 * (NUMREFLECTION - k +1)));
			pointmatrix[indice] = maxp;
			
		}//end for(int k=0; k<NUMREFLECTION;k++)


}//fine metodo reflection





__global__ void setup_kernel( hiprandState *state, unsigned long seed)
{
    int id = threadIdx.x;
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset = x + y * blockDim.x * gridDim.x;

	seed = seed + offset;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__global__ void kernel(Sphere *s, unsigned char *ptr, int *matrix, hiprandState* globalState, Point *pointmatrix) {

	
	Point points[SPHERES];

	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset = (DIM * y)+x;

	int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
	int start = 0;
	int end = DIM;
	
	int count=0;


	for(int i=0;i<1;i++){
		
		float rnd_number = hiprand_uniform(&localState);
		int rnd_integer_from_A_to_B_X = start + rnd_number * (end-start);
		rnd_number = rnd_number = hiprand_uniform(&localState);
		int rnd_integer_from_A_to_B_Y = start + rnd_number * (end-start);
		rnd_number = rnd_number = hiprand_uniform(&localState);
		int rnd_integer_from_A_to_B_Z = start + rnd_number * (end-start);
		globalState[ind] = localState; 
		int RX = rnd_integer_from_A_to_B_X;
		int RY = rnd_integer_from_A_to_B_Y;
		int RZ = rnd_integer_from_A_to_B_Z;

		double dirx;
		double diry;
		double dirz;

		dirx=RX-LIGHTX;
		diry=RY-LIGHTY;
		dirz=RZ-LIGHTZ;

		for (int j = 0; j<SPHERES; j++) {
			
				
				Point p;
				p.x=-1;
				p.y=-1;
				p.z=-1;
				p.d=-1;
				p.j=-1;

				float A = pow(dirx,2.0)+pow(diry,2.0)+pow(dirz,2.0);
				float B = 2.0 * (((LIGHTX-s[j].x)*dirx + (LIGHTY-s[j].y)*diry + (LIGHTZ - s[j].z )*dirz));
				float C = (pow((LIGHTX-s[j].x),2) + pow((LIGHTY-s[j].y),2) + pow((LIGHTZ-s[j].z),2)) - pow(s[j].radius,2);
				float D = B*B-4*A*C;
				

				if(D > 0.0){
					

					double t0 = ((-B) + sqrtf(D))/(2*A);
					double t1 = ((-B) - sqrtf(D))/(2*A);

					double ix0 = LIGHTX + (t0*dirx);
					double iy0 = LIGHTY + (t0*diry);
					double iz0 = LIGHTZ + (t0*dirz);

					double ix1 = LIGHTX + (t1*dirx);
					double iy1 = LIGHTY + (t1*diry);
					double iz1 = LIGHTZ + (t1*dirz);					

					double d0 = sqrt(pow((ix0-LIGHTX),2)+pow((iy0-LIGHTY),2)+pow((iz0-LIGHTZ),2));
					double d1 = sqrt(pow((ix1-LIGHTX),2)+pow((iy1-LIGHTY),2)+pow((iz1-LIGHTZ),2));

					int ix;
					int iy;
					int iz;
					double d;

					if(t0>0 && t1>0){
						if(t0<t1){
							ix = ix0;
							iy = iy0;
							iz = iz0;
							d=t0;
						}else{
							ix = ix1;
							iy = iy1;
							iz = iz1;
							d=t1;
						}
					}else{
						if(t0<0 && t1>0){
							ix = ix1;
							iy = iy1;
							iz = iz1;
							d=t1;
						}else{
							if(t1<0 && t0>0){
								ix = ix0;
								iy = iy0;
								iz = iz0;
								d=t0;
							}
						}
					}

					if(ix<0 || iy<0 || iz <0 || ix>DIM || iy>DIM || iz > DIM)
						continue;

					p.x=ix;
					p.y=iy;
					p.z=iz;
					p.d=d;
					p.j=j;
					points[j]=p;


					count++;
					

				}
				if(D < 0.0){
					points[j]=p;
				}

			}

			Point maxp;
			maxp.d=4294967295;

			for (int j = 0; j<SPHERES; j++) {


				if(points[j].x == -1 || (points[j].x == 0 && points[j].y == 0 && points[j].z == 0 && points[j].d == 0) || points[j].d<0 )
					continue;

				

				if(maxp.d>points[j].d){
					maxp=points[j];
				}

			}

			if(maxp.d==4294967295)
				continue;

			int indice = (DIM * maxp.y)+maxp.x;
			atomicAdd(&matrix[indice],(matrix[indice] +=1 * NUMREFLECTION));
			pointmatrix[indice] = maxp;
			
		///////////////////RIMBALZI

			reflection( maxp, dirx,  diry, dirz,  s,ptr,matrix, pointmatrix);



			
		__syncthreads();


					int countAVG = 0;
					int radiusAVG = 2;
					
					countAVG += controlSide(indice,matrix,radiusAVG);
				
					for(int k = 1; k <= radiusAVG;k++){

						if( ( indice - (DIM * radiusAVG)) >= 0 && (indice + (DIM * radiusAVG) ) <= DIM*DIM){
						//controllo sopra e sotto la matrice
			
						for(int i = 1; i < radiusAVG; i++){

							int up = i * DIM;
							int down = -i * DIM;

							 countAVG += matrix[indice + up];
							
							countAVG  += matrix[indice + down];
							


							countAVG += controlSide(indice + up,matrix,radiusAVG);
				
							countAVG += controlSide(indice + down,matrix,radiusAVG);

						}


						}
		
				}
			//countAVG += matrix[indice];// conto il pixel centrale
			int centro = matrix[indice];
			Point center = pointmatrix[indice];
			Sphere mysphere = s[center.j];


			float media = countAVG/((radiusAVG*2+1)*(radiusAVG*2+1));
			float scale = 1-(NUMREFLECTION/media)+0.4;
			if(scale<0){
				scale = 0.4;
			}
			if(scale>1){
				scale=1;
			}
			
			ptr[indice * 4 + 0] = scale * (mysphere.r);
			ptr[indice * 4 + 1] = scale * (mysphere.g);
			ptr[indice * 4 + 2] = scale * (mysphere.b);
			ptr[indice * 4 + 3] = scale * 255;
			


	}

}//fine metodo kernel







// globals needed by the update routine
struct DataBlock {
	unsigned char   *dev_bitmap;
	Sphere          *s;
};

//singola esecuzione del Photon Mapping
float PhotonMap(void){
	hipProfilerStart();
	DataBlock   data;
	CPUBitmap bitmap(DIM, DIM, &data);
	// capture the start time
	hipEvent_t     start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	unsigned char   *dev_bitmap;
	Sphere          *s;



	// allocate memory on the GPU for the output bitmap
	hipMalloc((void**)&dev_bitmap, bitmap.image_size());

	unsigned char* temp_bitmap= (unsigned char*)malloc(bitmap.image_size());

	for(int i=0;i<DIM*DIM;i++){
			temp_bitmap[i * 4 + 0] = 0;
			temp_bitmap[i * 4 + 1] = 0;
			temp_bitmap[i * 4 + 2] = 0;
			temp_bitmap[i * 4 + 3] = 255;
	}

	hipMemcpy(dev_bitmap, temp_bitmap, bitmap.image_size(), hipMemcpyHostToDevice);

// allocate memory for the Sphere dataset
	hipMalloc((void**)&s, sizeof(Sphere) * SPHERES);

	int* hostmatrix;
	int* devicematrix;

	

	hipMalloc((void**)&devicematrix, DIM*DIM*sizeof(int));
	hostmatrix = (int* )malloc(DIM*DIM*sizeof(int));

	for(int i = 0; i< DIM*DIM;i++){
	hostmatrix[i] = 0;
	}

	hipMemcpy(devicematrix ,hostmatrix, sizeof(int) * DIM * DIM, hipMemcpyHostToDevice);

	Point* pointmatrix, * fakematrix;
	hipMalloc((void**)&pointmatrix, DIM*DIM*sizeof(Point));
	fakematrix = (Point* )malloc(DIM*DIM*sizeof(Point));
	Point p;
	p.x=-1;
	p.y=-1;
	p.z=-1;
	p.d=-1;
	p.j=-1;

	for(int i = 0; i< DIM*DIM;i++){
		fakematrix[i] = p;
	}

	hipMemcpy(pointmatrix ,fakematrix, sizeof(Point) * DIM * DIM, hipMemcpyHostToDevice);
	free(fakematrix);

	// allocate temp memory, initialize it, copy to
	// memory on the GPU, then free our temp memory
	srand(time(NULL));

	Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
	for (int i = 0; i<SPHERES; i++) {
		
		temp_s[i].r = rand()%256;
		temp_s[i].g = rand()%256;
		temp_s[i].b = rand()%256;

		temp_s[i].radius = 30+(rand()%41);
		temp_s[i].x = rand()%DIM;
		temp_s[i].y =rand()%DIM;
		temp_s[i].z = rand()%DIM/6;
		//printf("Sfera %d : x=%f y=%f z=%f radius=%f\n", i, temp_s[i].x,temp_s[i].y,temp_s[i].z,temp_s[i].radius);
	}

	hipMemcpy(s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);
	

	// generate a bitmap from our sphere data
	dim3    grids(DIMTH / 16, DIMTH / 16);
	dim3    threads(16, 16);

	int N = DIMTH*DIMTH;
	hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );
    
    // setup seeds
    setup_kernel <<< grids, threads >>> ( devStates, time(NULL) );

	kernel << <grids, threads >> >(s, dev_bitmap, devicematrix,devStates,pointmatrix);
	
	hipProfilerStop();



	// copy our bitmap back from the GPU for display
	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
	hipMemcpy(hostmatrix, devicematrix, DIM*DIM*sizeof(int), hipMemcpyDeviceToHost);

	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float   elapsedTime;
	hipEventElapsedTime(&elapsedTime,start, stop);
	printf("Time to generate:  %3.1f ms\n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	free(temp_s);
	hipFree(dev_bitmap);
	hipFree(s);
	hipFree(devicematrix);
	hipFree(pointmatrix);
	hipFree(devStates);

	//bitmap.display_and_exit();
	return elapsedTime;
}

int main(int argc, char* argv[]) {

	ofstream myfile;
	char buffer[100];
	float realTime;
	float expectedTime=0.0;

	std::string name="PhotonMapping_S";
	name=name+itoa(SPHERES,buffer,10);
	name=name+"_D"+itoa(DIM,buffer,10);
	name=name+"_TH"+itoa(DIMTH,buffer,10);
	name=name+"_T"+itoa(NUMTEST,buffer,10)+".txt";
	myfile.open(name);

	for(int i=0;i<NUMTEST;i++){
		realTime=PhotonMap();
		expectedTime+=realTime;
		sprintf(buffer, "Experiment %d Time:  %3.1f ms\n", i+1, realTime);
		myfile<<buffer;
	}
	expectedTime=expectedTime/NUMTEST;
	//printf("Expected Time:  %3.1f ms\n", expectedTime);

	
	sprintf(buffer, "Mean Time:  %3.1f ms\n", expectedTime);
	myfile<<buffer;
	myfile.close();
}
